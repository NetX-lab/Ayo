#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include "attention_dtypes.h"
#include "attention_utils.cuh"

#include <algorithm>

#define WARP_SIZE 32
#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVIDE_ROUND_UP(a, b) (((a) + (b) - 1) / (b))


namespace vllm {
	
}